/* SorensonPar.cu
   Parallel Implementation of Algorithm 4.1
   as discussed in Sorenson and Parberry's
   1994 paper "Two Fast Parallel Prime Number
   Sieves".

   Authors:
   Daniel Anzaldo
   David Frank
   Antonio Lanfranchi
*/

// Visual Studio Dependencies (Can be commented out)
#include "hip/hip_runtime.h"


// C dependencies
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
using namespace std;

// C++ dependencies
#include <algorithm>

typedef unsigned long long big;

// GLOBAL VARIABLES--------------------------------------

bool * S;	// Global shared bit array of numbers up to N

/* These are for tracking time */
struct timezone myTimezone;	
struct timeval startTime, endTime;

// HOST FUNCTION HEADERS---------------------------------

/*	EratosthenesSieve
	HELPER: for Algorithm 4.1 Sequential Portion
	The most basic form of generating primes.
*/
void EratosthenesSieveNaive(big n);

/* Set a checkpoint and show the total running time in seconds */
double report_running_time(const char *arr);


/*	MAIN
	To run this add the ff. args:
	1. N = the number up to which you're sieving
*/
int main(int argc, char **argv)
{
	big N = (big)strtoul(argv[1], NULL, 10);
	S = new bool[N]; //(bool*)malloc(N * sizeof(bool));

	printf("Find primes up to: %llu\n\n", N);
	
	/* start counting time */
	gettimeofday(&startTime, &myTimezone);

	/* sieve for primes up to N */
	EratosthenesSieveNaive(N);

	/* check the total running time */ 
	report_running_time("Algorithm 4.1 CPU");

	// Display the primes.
	for (int i = 0; i < N; i++)
		if (S[i]) printf("%llu ", i);

	delete[] S;
    return 0;
}


// HOST FUNCTION DEFINITIONS-----------------------------

void EratosthenesSieveNaive(big n)
{
	// 0 and 1 are non-primes.
	S[0] = S[1] = false;
	for (big i = 2; i < n; i++)
		S[i] = true;

	// Simple Sieving Operation.
	for (big i = 2; i < n; i++)
		if (S[i])
		{
			int j;
			for (j = i*i; j < n; j += i)
				S[j] = false;
		}
}

/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time(const char *arr) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &myTimezone);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for %s: %ld.%06ld sec\n\n", arr, sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}
