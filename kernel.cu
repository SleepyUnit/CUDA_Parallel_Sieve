/* SorensonPar.cu
   Parallel Implementation of Algorithm 4.1
   as discussed in Sorenson and Parberry's
   1994 paper "Two Fast Parallel Prime Number
   Sieves".

   Authors:
   Daniel Anzaldo
   David Frank
   Antonio Lanfranchi
*/

// Visual Studio Dependencies (Can be commented out)
#include "hip/hip_runtime.h"


// C dependencies
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

// C++ dependencies
#include <algorithm>

typedef unsigned long long big;

// GLOBAL VARIABLES--------------------------------------
bool * S;	// Global shared bit array of numbers up to N
int P;		// Global number of processors

bool check_cuda_status = false; // turn to false when running on circe

/* These are for tracking time */
struct timezone myTimezone;	
struct timeval startTime, endTime;

// HOST FUNCTION HEADERS---------------------------------

/*	EratosthenesSieve
	HELPER: for Algorithm 4.1 Sequential Portion
	The most basic form of generating primes.
	Used to help find the first k primes.
	Returns the k-th prime.
*/
void EratosthenesSieve(big n);

/*	Algorithm 4.1 Sequential Portion
	Running Time: O(sqrt(n))
	Space: O(sqrt(n)) up to O(sqrt(n)/log log n)
*/
hipError_t algorithm4_1(big n);

/*	Algorithm 4.1 Helper: Parallel Sieve
	All CUDA-related functionality goes here.
	This code will change for different kernel versions.
*/
hipError_t parallelSieve(big n, big range);

/* Frees the memory allocated on the device and returns any errors*/
hipError_t cleanup(bool *d_S, hipError_t cudaStatus);

/* Set a checkpoint and show the total running time in seconds */
double report_running_time(const char *arr);


// DEVICE MATH FUNCTIONS---------------------------------

/*	sqrt_d
	Device version of the Square Root Function
	Babylonian Method
*/
__device__ big sqrt_d(big a)
{
   big root = a/2;
   
#pragma unroll
   for (big n = 0; n < 10; n++)
   {
      root = 0.5 * (root + (a/root));
   }
   
   return root;
}

__device__ big min_d(big a, big b)
{
   return (a < b) ? a : b;
}

__device__ big max_d(big a, big b)
{
	return (a > b) ? a : b;
}


// ALGORITHM 4.1 KERNEL VERSIONS-------------------------
/*	
	Parallel Sieve Kernel
*/
__global__ void parallelSieveKernel(
	big n, big range, bool *d_S)
{
	// Shared memory use for S in range of thread
	__shared__ bool primes[48000];

	if (threadIdx.x == 0)
	{
		for (int i=0; i<=48000; i++)
			primes[i] = d_S[i];
	}

	// Sync after initializing
	__syncthreads();

	// Variables
	int j;
	big f;

	// Thread id
	big id = threadIdx.x + blockIdx.x * blockDim.x;

	// Generate required values
	big sqrt_N = sqrt_d(n);

	// Find left and right range
	big L = range * id + sqrt_N;
	big R = range + L;

	// Sieve
	for (j = 0; j < sqrt_N; j++)
	{
		// For each prime number
		if (primes[j])
		{
			// Calculate smallest f
			f = L / j;

			while (j * f <= R)
			{
				d_S[j * f] = false;
				f++;
			}
		}
	}

	return;
}

/*	MAIN
	To run this add the ff. args:
	1. N = the number up to which you're sieving
*/
int main(int argc, char **argv)
{
	big N = (big)strtoull(argv[1], NULL, 10);
	S = new bool[N]; //(bool*)malloc(N * sizeof(bool));

	if (sqrtl(N) > 48000)
	{
		printf("Argument is too large, maximum argument is 2,304,000,000\n");
		return 0;
	}

	printf("Find primes up to: %llu\n\n", N);
	
	/* start counting time */
	gettimeofday(&startTime, &myTimezone);

	hipError_t x = algorithm4_1(N);

	/* check the total running time */ 
	report_running_time("CUDA Parallel Sieve");

	if (check_cuda_status)
	{
		if (x != hipSuccess) {
			printf("CUDA Parallel Sieve failed to execute!");
			return 1;
		}
	}

	// Display the primes.
	for (int i = 0; i < N; i++)
		if (S[i]) printf("%llu ", i);

	delete[] S;
    return 0;
}


// HOST FUNCTION DEFINITIONS-----------------------------

void EratosthenesSieve(big n)
{
	// 0 and 1 are non-primes.
	S[0] = S[1] = false;
	for (big i = 2; i < n; i++)
		S[i] = true;

	// Simple Sieving Operation up to square root of n
	for (big i = 2; i < (big)sqrtl(n); i++)
		if (S[i])
		{
			int j;
			for (j = i*i; j < n; j += i)
				S[j] = false;
		}
      
   return;
}

hipError_t algorithm4_1(big n)
{
	/* VARIABLES */
	big range;
	big sqrt_N = (big)sqrtl((long double)n);

	/* Find the first k primes up to sqrt(N) */
	EratosthenesSieve(n);

	/* Delta = ceil(n/p) */
	range = (big)ceill(n / (long double)P);

	/* PARALLEL PART */
	hipError_t parallelStatus = parallelSieve(n, range);
	if (check_cuda_status)
	{
		if (parallelStatus != hipSuccess) {
			fprintf(stderr, "parallelSieve() failed!");
		}
	}

	return parallelStatus;
}

hipError_t parallelSieve(big n, big range)
{
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/* The Number Field S
	   will be migrated to GLOBAL memory
	   OPTIMIZATION: ranges will be migrated to SHARED memory
	   OPTIMIZATION: [0, sqrt(n)] will be migrated to CONSTANT memory
	*/
	bool * d_S = NULL;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (check_cuda_status)
	{
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
			return cudaStatus;
		}
	}

	// Measure start time for CUDA portion
	hipEventRecord(start, 0);

	// CUDA Memory Allocations.
	cudaStatus = hipMalloc((void**)&d_S, n * sizeof(bool));
	if (check_cuda_status)
	{
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed on number field S!\n");
			return cleanup(d_S, cudaStatus);
		}
	}

	//  cudaMemCpy -> Device
	cudaStatus = hipMemcpy(d_S, S, n * sizeof(bool), hipMemcpyHostToDevice);
	if (check_cuda_status)
	{
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed! S->d_S.\n");
			return cleanup(d_S, cudaStatus);
		}
	}

	// Kernel Call
	dim3 gridSize(ceill(ceill(sqrt(n))/256), 1, 1);
	dim3 blockSize(256, 1, 1);

	//parallelSieveKernel<<<gridSize, blockSize>>>(n, k, m, wheel, range, d_S);
	parallelSieveKernel<<<gridSize, blockSize>>>(n, range, d_S);

	cudaStatus = hipGetLastError();
	if (check_cuda_status)
	{
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "parallelSieveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cleanup(d_S, cudaStatus);
		}
	}

	cudaStatus = hipDeviceSynchronize();
	if (check_cuda_status)
	{
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return cleanup(d_S, cudaStatus);
		}
	}

	// cudaMemCpy -> Host
	cudaStatus = hipMemcpy(S, d_S, n * sizeof(bool), hipMemcpyDeviceToHost);
	if (check_cuda_status)
	{
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed! d_S->S.\n");
			return cleanup(d_S, cudaStatus);
		}
	}

	// Measure stop time for CUDA portion
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate: %0.5f ms\n", elapsedTime);

	// cudaFree
	return cleanup(d_S, cudaStatus);
}

hipError_t cleanup(bool *d_S, hipError_t cudaStatus)
{
	hipFree(d_S);
	return cudaStatus;
}

/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time(const char *arr) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &myTimezone);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for %s: %ld.%06ld sec\n\n", arr, sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}
